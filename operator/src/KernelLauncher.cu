#include <KernelLauncher.cuh>

template <typename... Args>
KernelLauncher<Args...>::KernelLauncher() {};

template <typename... Args>
KernelLauncher<Args...>::KernelLauncher(
void (*fwd_kernel)(complex_vector* __restrict__, complex_vector* __restrict__, Args...), 
void (*adj_kernel)(complex_vector* __restrict__, complex_vector* __restrict__, Args...), 
dim3 grid, dim3 block, hipStream_t stream) : 
_grid_(grid), _block_(block), _fwd_kernel_(fwd_kernel), _adj_kernel_(adj_kernel), _stream_(stream) {};

template <typename... Args>
KernelLauncher<Args...>::KernelLauncher(
void (*fwd_kernel)(complex_vector* __restrict__, complex_vector* __restrict__, Args...), 
dim3 grid, dim3 block, hipStream_t stream) :
 _grid_(grid), _block_(block), _fwd_kernel_(fwd_kernel), _stream_(stream) {};

template <typename... Args>
KernelLauncher<Args...>::~KernelLauncher() {};

template <typename... Args>
void KernelLauncher<Args...>::run_fwd(complex_vector* __restrict__ model, complex_vector* __restrict__ data, Args... args) {
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(model, sizeof(complex_vector), hipCpuDeviceId, _stream_)); 
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(data, sizeof(complex_vector), hipCpuDeviceId, _stream_)); 
    _fwd_kernel_<<<_grid_, _block_, 0, _stream_>>>(model, data, args...);
    CHECK_CUDA_ERROR( hipPeekAtLastError() );
    // CHECK_CUDA_ERROR(hipStreamSynchronize(_stream_));
  };

template <typename... Args>
void KernelLauncher<Args...>::run_adj(complex_vector* __restrict__ model, complex_vector* __restrict__ data, Args... args) {
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(model, sizeof(complex_vector), hipCpuDeviceId, _stream_)); 
    CHECK_CUDA_ERROR(hipMemPrefetchAsync(data, sizeof(complex_vector), hipCpuDeviceId, _stream_)); 
    _adj_kernel_<<<_grid_, _block_, 0, _stream_>>>(model, data, args...);
    CHECK_CUDA_ERROR( hipPeekAtLastError() );
    // CHECK_CUDA_ERROR(hipStreamSynchronize(_stream_));
  };