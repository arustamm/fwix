#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <iostream>

__device__ void CB_ortho(void *dataOut, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr) {
  int* n = static_cast<int*>(callerInfo);
  float norm_factor = rsqrtf(static_cast<float>(n[0]));
  ((hipfftComplex*)dataOut)[offset] = hipCmulf(element, make_hipFloatComplex(norm_factor, 0.0f));
}
__device__ hipfftCallbackStoreC d_storeCallbackPtr = CB_ortho;

hipfftCallbackStoreC get_host_callback_ptr() {
  hipfftCallbackStoreC h_storeCallbackPtr;
  hipMemcpyFromSymbol(&h_storeCallbackPtr, HIP_SYMBOL(d_storeCallbackPtr), sizeof(h_storeCallbackPtr));
  return h_storeCallbackPtr;
}

