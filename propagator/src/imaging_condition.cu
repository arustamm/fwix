#include "hip/hip_runtime.h"
#include <complex_vector.h>
#include <prop_kernels.cuh>
#include <hip/hip_complex.h>
#include <KernelLauncher.cuh>
#include <KernelLauncher.cu>

#define TILE_DIM 16

__global__ void ic_fwd(
    complex_vector* __restrict__ model,  
    complex_vector* __restrict__ data,
    const complex_vector* __restrict__ bg_wfld
) {
    // Here the model is not padded while the data is padded.
    const int nx = model->n[0];
    const int ny = model->n[1];
    
    const int NX = data->n[0];
    const int NY = data->n[1];
    const int NW = data->n[2];
    const int NS = data->n[3];
    
    int ix0 = threadIdx.x + blockDim.x * blockIdx.x;
    int iy0 = threadIdx.y + blockDim.y * blockIdx.y;
    int iws0 = threadIdx.z + blockDim.z * blockIdx.z;

    int jx = blockDim.x * gridDim.x;
    int jy = blockDim.y * gridDim.y;
    int jws = blockDim.z * gridDim.z;

    const int dims[] = {NS, NW, NY, NX};
    const int im_dims[] = {NW, ny, nx};

    for (int iws = iws0; iws < NW*NS; iws += jws) {
        int iw = iws / NS;  // Frequency index
        int is = iws % NS; // Source index

        // Loop over spatial points in the receiver wavefield
      for (int iy = iy0; iy < ny; iy += jy) {
        for (int ix = ix0; ix < nx; ix += jx) {

          int im_ind[] = {iw, iy, ix};
          size_t flat_ind = ND_TO_FLAT(im_ind, im_dims);
          hipFloatComplex img_val = model->mat[flat_ind];

          int nd_ind[] = {is, iw, iy, ix};
          flat_ind = ND_TO_FLAT(nd_ind, dims);

          // Get values
          hipFloatComplex src_val = bg_wfld->mat[flat_ind];
          data->mat[flat_ind] = hipCaddf(data->mat[flat_ind], hipCmulf(src_val, img_val));
      }
    }
  }
}

__global__ void ic_adj(
    complex_vector* __restrict__ model,  
    complex_vector* __restrict__ data,
    const complex_vector* __restrict__ bg_wfld
) {
  const int nx = model->n[0];
  const int ny = model->n[1];
  
  const int NX = data->n[0];
  const int NY = data->n[1];
  const int NW = data->n[2];
  const int NS = data->n[3];
  
  int ix0 = threadIdx.x + blockDim.x * blockIdx.x;
  int iy0 = threadIdx.y + blockDim.y * blockIdx.y;
  int iws0 = threadIdx.z + blockDim.z * blockIdx.z;

  int jx = blockDim.x * gridDim.x;
  int jy = blockDim.y * gridDim.y;
  int jws = blockDim.z * gridDim.z;

  const int dims[] = {NS, NW, NY, NX};
  const int im_dims[] = {NW, ny, nx};

  for (int iws = iws0; iws < NW*NS; iws += jws) {
      int iw = iws / NS;  // Frequency index
      int is = iws % NS; // Source index

      // Loop over spatial points in the receiver wavefield
    for (int iy = iy0; iy < ny; iy += jy) {
      for (int ix = ix0; ix < nx; ix += jx) {

        int nd_ind[] = {is, iw, iy, ix};
        size_t flat_ind = ND_TO_FLAT(nd_ind, dims);

        // Get values
        hipFloatComplex src_val = bg_wfld->mat[flat_ind];
        hipFloatComplex rec_val = data->mat[flat_ind];
        
        // Adjoint: conj(source) * receiver
        hipFloatComplex contribution = hipCmulf(hipConjf(src_val), rec_val);

        // Atomic add to model (multiple threads will write to same (iw,iy,ix) location)
        int im_ind[] = {iw, iy, ix};
        size_t img_flat_ind = ND_TO_FLAT(im_ind, im_dims);
        
        atomicAdd(&(model->mat[img_flat_ind].x), hipCrealf(contribution));
        atomicAdd(&(model->mat[img_flat_ind].y), hipCimagf(contribution));
      }
    }
  }
}



