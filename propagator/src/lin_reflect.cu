#include "hip/hip_runtime.h"
#include <complex_vector.h>
#include <prop_kernels.cuh>
#include <hip/hip_complex.h>
#include <math.h>
#include <KernelLauncher.cuh>
#include <KernelLauncher.cu>

// Combined linearized reflection forward operator
__global__ void drefl_forward(
  complex_vector* __restrict__ model,  
  complex_vector* __restrict__ data,
  const complex_vector* __restrict__ model_slow,
  const complex_vector* __restrict__ model_den) {

  int NX = model->n[0];
  int NY = model->n[1];
  int NW = model->n[2];

  hipFloatComplex* dslow = model->mat; // Slowness perturbation
  hipFloatComplex* dden = model->mat + 2 * NX * NY * NW; // Density perturbation
  
  int data_dims[] = {NW, NY, NX};           // 3D data array
  int slice_dims[] = {2, NW, NY, NX};       // Background model slices

  int ix0 = threadIdx.x + blockDim.x * blockIdx.x;
  int iy0 = threadIdx.y + blockDim.y * blockIdx.y;
  int iw0 = threadIdx.z + blockDim.z * blockIdx.z;

  int jx = blockDim.x * gridDim.x;
  int jy = blockDim.y * gridDim.y;
  int jw = blockDim.z * gridDim.z;

  for (int iw = iw0; iw < NW; iw += jw) {
    for (int iy = iy0; iy < NY; iy += jy) {
      for (int ix = ix0; ix < NX; ix += jx) {

        int slice_nd_ind[] = {0, iw, iy, ix};
        size_t slice_ind_0 = ND_TO_FLAT(slice_nd_ind, slice_dims);
        size_t slice_ind_1 = slice_ind_0 + NW * NX * NY;

        int data_nd_ind[] = {iw, iy, ix};
        size_t data_ind = ND_TO_FLAT(data_nd_ind, data_dims);

        // Compute coefficients on-the-fly
        hipFloatComplex sqrt_s0 = csqrtf(model_slow->mat[slice_ind_0]);
        hipFloatComplex sqrt_s1 = csqrtf(model_slow->mat[slice_ind_1]);
        hipFloatComplex rho0 = model_den->mat[slice_ind_0];
        hipFloatComplex rho1 = model_den->mat[slice_ind_1];

        // Denominator: (rho1*sqrt(s0) + rho0*sqrt(s1))
        hipFloatComplex denom = hipCaddf(
            hipCmulf(rho1, sqrt_s0),
            hipCmulf(rho0, sqrt_s1)
        );

        hipFloatComplex inv_denom = hipCdivf(make_hipFloatComplex(1.0f, 0.0f), denom);
        hipFloatComplex inv_denom_sq = hipCmulf(inv_denom, inv_denom);

        // Coefficients computed directly
        hipFloatComplex s1s2sq = hipCmulf(hipCmulf(rho0, rho1), inv_denom_sq);
        hipFloatComplex r1r2sq = hipCmulf(
            hipCmulf(make_hipFloatComplex(2.0f, 0.0f), hipCmulf(sqrt_s0, sqrt_s1)), 
            inv_denom_sq
        );

        hipFloatComplex result = make_hipFloatComplex(0.0f, 0.0f);

        // Slowness perturbation contribution
        hipFloatComplex c = hipCdivf(sqrt_s1, sqrt_s0);
        hipFloatComplex sqrt_c = csqrtf(c);
        hipFloatComplex inv_sqrt_c = hipCdivf(make_hipFloatComplex(1.0f, 0.0f), sqrt_c);

        hipFloatComplex a = hipCmulf(sqrt_c, s1s2sq);
        hipFloatComplex b = hipCmulf(inv_sqrt_c, s1s2sq);

        result = hipCaddf(result, hipCsubf(
            hipCmulf(a, dslow[slice_ind_0]),
            hipCmulf(b, dslow[slice_ind_1])
        ));

        // Density perturbation contribution
        a = hipCmulf(rho0, r1r2sq);
        b = hipCmulf(rho1, r1r2sq);

        result = hipCaddf(result, hipCsubf(
            hipCmulf(a, dden[slice_ind_1]),
            hipCmulf(b, dden[slice_ind_0])
        ));

        data->mat[data_ind] = hipCaddf(data->mat[data_ind], result);
      }
    }
  }
}

// Combined linearized reflection adjoint operator
__global__ void drefl_adjoint(
    complex_vector* __restrict__ model,
    complex_vector* __restrict__ data,
    const complex_vector* __restrict__ model_slow,
    const complex_vector* __restrict__ model_den) {

  int NX = model->n[0];
  int NY = model->n[1];
  int NW = model->n[2];

  hipFloatComplex* dslow = model->mat; // Slowness perturbation
  hipFloatComplex* dden = model->mat + 2 * NX * NY * NW; // Density perturbation
  
  int data_dims[] = {NW, NY, NX};           // 3D data array
  int slice_dims[] = {2, NW, NY, NX};       // Background model slices

  int ix0 = threadIdx.x + blockDim.x * blockIdx.x;
  int iy0 = threadIdx.y + blockDim.y * blockIdx.y;
  int iw0 = threadIdx.z + blockDim.z * blockIdx.z;

  int jx = blockDim.x * gridDim.x;
  int jy = blockDim.y * gridDim.y;
  int jw = blockDim.z * gridDim.z;

  for (int iw = iw0; iw < NW; iw += jw) {
    for (int iy = iy0; iy < NY; iy += jy) {
      for (int ix = ix0; ix < NX; ix += jx) {

        int slice_nd_ind[] = {0, iw, iy, ix};
        size_t slice_ind_0 = ND_TO_FLAT(slice_nd_ind, slice_dims);
        size_t slice_ind_1 = slice_ind_0 + NW * NX * NY;

        int data_nd_ind[] = {iw, iy, ix};
        size_t data_ind = ND_TO_FLAT(data_nd_ind, data_dims);

        hipFloatComplex data_val = data->mat[data_ind];

        // Compute coefficients on-the-fly (same as forward)
        hipFloatComplex sqrt_s0 = csqrtf(model_slow->mat[slice_ind_0]);
        hipFloatComplex sqrt_s1 = csqrtf(model_slow->mat[slice_ind_1]);
        hipFloatComplex rho0 = model_den->mat[slice_ind_0];
        hipFloatComplex rho1 = model_den->mat[slice_ind_1];

        hipFloatComplex denom = hipCaddf(
            hipCmulf(rho1, sqrt_s0),
            hipCmulf(rho0, sqrt_s1)
        );

        hipFloatComplex inv_denom = hipCdivf(make_hipFloatComplex(1.0f, 0.0f), denom);
        hipFloatComplex inv_denom_sq = hipCmulf(inv_denom, inv_denom);

        hipFloatComplex s1s2sq = hipCmulf(hipCmulf(rho0, rho1), inv_denom_sq);
        hipFloatComplex r1r2sq = hipCmulf(
            hipCmulf(make_hipFloatComplex(2.0f, 0.0f), hipCmulf(sqrt_s0, sqrt_s1)), 
            inv_denom_sq
        );

        // Slowness perturbation adjoint
        hipFloatComplex c = hipCdivf(sqrt_s1, sqrt_s0);
        hipFloatComplex sqrt_c = csqrtf(c);
        hipFloatComplex inv_sqrt_c = hipCdivf(make_hipFloatComplex(1.0f, 0.0f), sqrt_c);

        hipFloatComplex a = hipConjf(hipCmulf(sqrt_c, s1s2sq));
        hipFloatComplex b = hipConjf(hipCmulf(inv_sqrt_c, s1s2sq));

        hipFloatComplex contrib_slow_0 = hipCmulf(a, data_val);
        hipFloatComplex contrib_slow_1 = hipCmulf(hipCmulf(make_hipFloatComplex(-1.0f, 0.0f), b), data_val);

        atomicAdd(&(dslow[slice_ind_0].x), contrib_slow_0.x);
        atomicAdd(&(dslow[slice_ind_0].y), contrib_slow_0.y);
        atomicAdd(&(dslow[slice_ind_1].x), contrib_slow_1.x);
        atomicAdd(&(dslow[slice_ind_1].y), contrib_slow_1.y);

        // Density perturbation adjoint
        a = hipConjf(hipCmulf(rho0, r1r2sq));
        b = hipConjf(hipCmulf(rho1, r1r2sq));

        hipFloatComplex contrib_den_1 = hipCmulf(a, data_val);
        hipFloatComplex contrib_den_0 = hipCmulf(hipCmulf(make_hipFloatComplex(-1.0f, 0.0f), b), data_val);

        atomicAdd(&(dden[slice_ind_1].x), contrib_den_1.x);
        atomicAdd(&(dden[slice_ind_1].y), contrib_den_1.y);
        atomicAdd(&(dden[slice_ind_0].x), contrib_den_0.x);
        atomicAdd(&(dden[slice_ind_0].y), contrib_den_0.y);
      }
    }
  }
}

