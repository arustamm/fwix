#include "hip/hip_runtime.h"
#include <complex_vector.h>
#include <prop_kernels.cuh>
#include <hip/hip_complex.h>
#include <KernelLauncher.cuh>
#include <KernelLauncher.cu>

template class KernelLauncher<float*, float*, float*, hipFloatComplex*, float, float>;

__global__ void ps_forward(complex_vector* __restrict__ model, complex_vector* __restrict__ data, 
  float* __restrict__  w2, float* __restrict__  kx, float* __restrict__  ky, hipFloatComplex* __restrict__ slow_ref, float dz, float eps) {

  float a, b, c, re, im;
  size_t flat_ind;
  int NX = model->n[0];
  int NY = model->n[1];
  int NW = model->n[2];
  int NS = model->n[3];
  int dims[] = {NS, NW, NY, NX};

  int ix0 = threadIdx.x + blockDim.x*blockIdx.x;
  int iy0 = threadIdx.y + blockDim.y*blockIdx.y;
  int iw0 = threadIdx.z + blockDim.z*blockIdx.z;

  int jx = blockDim.x * gridDim.x;
  int jy = blockDim.y * gridDim.y;
  int jw = blockDim.z * gridDim.z;

  for (int is=0; is < NS; ++is) {

  for (int iw=iw0; iw < NW; iw += jw) {
    float sre = hipCrealf(slow_ref[iw]);
    float sim = hipCimagf(slow_ref[iw]);
    for (int iy=iy0; iy < NY; iy += jy) {
      for (int ix=ix0; ix < NX; ix += jx) {
        a = w2[iw]*sre - (kx[ix]*kx[ix] + ky[iy]*ky[iy]);
        b = w2[iw]*(sim-eps*sre);
        c = sqrtf(a*a + b*b);
        if (b <= 0) {
          re = sqrtf((c+a)/2);
          im = -sqrtf((c-a)/2);
        }
        else {
          re = -sqrtf((c+a)/2);
				  im = -sqrtf((c-a)/2);
        }
        
        // convert 4d index to flat index

          int nd_ind[] = {is, iw, iy, ix};
          flat_ind = ND_TO_FLAT(nd_ind, dims);

          float att = exp(im*dz);
          float coss = cos(re*dz);
          float sinn = sin(re*dz);

          float mre = hipCrealf(model->mat[flat_ind]);
          float mim = hipCimagf(model->mat[flat_ind]);

          re = att * (mre * coss + mim * sinn);
          im = att * (-mre * sinn + mim * coss);

          data->mat[flat_ind] = hipCaddf(data->mat[flat_ind], make_hipFloatComplex(re, im)); 
        }
      }
    }
  }
};

__global__ void ps_adjoint(complex_vector* __restrict__ model, complex_vector* __restrict__ data, float* w2, float* kx, float* ky, hipFloatComplex* slow_ref, float dz, float eps) {
  
  float a, b, c, re, im;
  size_t flat_ind;

  int NX = model->n[0];
  int NY = model->n[1];
  int NW = model->n[2];
  int NS = model->n[3];
  int dims[] = {NS, NW, NY, NX};

  int ix0 = threadIdx.x + blockDim.x*blockIdx.x;
  int iy0 = threadIdx.y + blockDim.y*blockIdx.y;
  int iw0 = threadIdx.z + blockDim.z*blockIdx.z;

  int jx = blockDim.x * gridDim.x;
  int jy = blockDim.y * gridDim.y;
  int jw = blockDim.z * gridDim.z;

  for (int is=0; is < NS; ++is) {

  for (int iw=iw0; iw < NW; iw += jw) {
    float sre = hipCrealf(slow_ref[iw]);
    float sim = hipCimagf(slow_ref[iw]);
    for (int iy=iy0; iy < NY; iy += jy) {
      for (int ix=ix0; ix < NX; ix += jx) {
        a = w2[iw]*sre - (kx[ix]*kx[ix] + ky[iy]*ky[iy]);
        b = w2[iw]*(sim-eps*sre);
        c = sqrtf(a*a + b*b);
        if (b <= 0) {
          re = sqrtf((c+a)/2);
          im = -sqrtf((c-a)/2);
        }
        else {
          re = -sqrtf((c+a)/2);
				  im = -sqrtf((c-a)/2);
        }

          // convert 4d index to flat index
          int nd_ind[] = {is, iw, iy, ix};
          flat_ind = ND_TO_FLAT(nd_ind, dims);

          float att = exp(im*dz);
          float coss = cos(re*dz);
          float sinn = sin(re*dz);

          float dre = hipCrealf(data->mat[flat_ind]);
          float dim = hipCimagf(data->mat[flat_ind]);

          re = att * (dre * coss - dim * sinn);
          im = att * (dre * sinn + dim * coss);

          model->mat[flat_ind] = hipCaddf(model->mat[flat_ind], make_hipFloatComplex(re, im));
        }
      }
    }
  }
};

__global__ void ps_inverse(complex_vector* __restrict__ model, complex_vector* __restrict__ data, float* w2, float* kx, float* ky, hipFloatComplex* slow_ref, float dz, float eps) {
  
  float a, b, c, re, im;
  size_t flat_ind;

  int NX = model->n[0];
  int NY = model->n[1];
  int NW = model->n[2];
  int NS = model->n[3];
  int dims[] = {NS, NW, NY, NX};

  int ix0 = threadIdx.x + blockDim.x*blockIdx.x;
  int iy0 = threadIdx.y + blockDim.y*blockIdx.y;
  int iw0 = threadIdx.z + blockDim.z*blockIdx.z;

  int jx = blockDim.x * gridDim.x;
  int jy = blockDim.y * gridDim.y;
  int jw = blockDim.z * gridDim.z;

  for (int is=0; is < NS; ++is) {

  for (int iw=iw0; iw < NW; iw += jw) {
    float sre = hipCrealf(slow_ref[iw]);
    float sim = hipCimagf(slow_ref[iw]);
    for (int iy=iy0; iy < NY; iy += jy) {
      for (int ix=ix0; ix < NX; ix += jx) {
        a = w2[iw]*sre - (kx[ix]*kx[ix] + ky[iy]*ky[iy]);
        b = w2[iw]*(sim-eps*sre);
        c = sqrtf(a*a + b*b);
        if (b <= 0) {
          re = sqrtf((c+a)/2);
          im = sqrtf((c-a)/2);
        }
        else {
          re = -sqrt((c+a)/2);
				  im = sqrt((c-a)/2);
        }

          // convert 4d index to flat index
          int nd_ind[] = {is, iw, iy, ix};
          flat_ind = ND_TO_FLAT(nd_ind, dims);

          float att = exp(im*dz);
          float coss = cos(re*dz);
          float sinn = sin(re*dz);

          float dre = hipCrealf(data->mat[flat_ind]);
          float dim = hipCimagf(data->mat[flat_ind]);

          re = att * (dre * coss - dim * sinn);
          im = att * (dre * sinn + dim * coss);

          model->mat[flat_ind] = hipCaddf(model->mat[flat_ind], make_hipFloatComplex(re, im));
        }
      }
    }
  }
}